
#include <hip/hip_runtime.h>
//#pragma warning(disable:4996)
//
//
//#include <device_functions.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include "common\book.h"
//
//
//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h> // math functions
//#include <string.h> // functions such as strcmp
//#include <iostream>
//#include <time.h>
//
//
///* a header file with common structure to for pgm file*/
//#include "src/pgm.h" // common header for common pgm functions
//
//
//#define MAX_FILE_LENGTH 256
//
////nucleia.pgm ascii
//#define  n1 511  //height of the image -1-rows
//#define  n2 511  //width of the image -1-columns
//
//
///* Definitions*/
//#define  h      0.01
//#define  tau    0.00025
//#define  sigma  0.000025
//#define  ni  100        /* a number of the iterations */
//#define  tol    1.0e-10   /* toleration for accuracy gauss-seidel */
//#define kk     10        /* a number of the diffusion steps */
//#define  iz     1         /* density of writing */
//#define  p     1   
//#define  omega_c     1.3      /*  SOR for the convolution   */
//#define  omega_s     1.4      /* SOR for the system */
//#define coeff 1   /* a konstant K of the function g(s) */
//
//
//double *partial_error;
//
//
//typedef  double pole1[n1 + 16 * p + 1][n2 + 16 * p + 1];
//typedef  int    pole2[n1 + 16 * p + 1][n2 + 16 * p + 1];
//
////pole--array
//static pole1 u, u0, uq, ap, ae,an,aw,as;
//static pole2 v;
//
//
//PGMImage *image; // image struct with the propertes: width, height, maxvalue
//char *name, *name_ending, *name_beg, *file_path, *rmode;
//int height = 0, width = 0; //height, width for input image
//int option, prd, off;
//FILE *in_file, *tmp_file;
//
///*functions for reading and writing*/
///*
//* reads the image data into a 2d array
//* img_array contains the image data
//*/
//
//void load_ascii()
//{
//	int ll, i, j;
//	//reading the image data - ascii_format
//	for (i = p; i <= height + p; i++)
//	{
//		for (j = p; j <= width + p; j++)
//		{
//			fscanf(in_file, "%d", &ll);
//			v[i][j] = ll;
//			u[i][j] = ll / 255.;
//		}
//	}
//	fclose(in_file);
//}
//
//void load_binary()
//{
//	int ll, i, j;
//	//reading the image data- binary format
//	for (i = p; i <= height + p; i++)
//	{
//		for (j = p; j <= width + p; j++)
//		{
//			ll = getc(in_file);
//			v[i][j] = ll;
//			u[i][j] = ll / 255.;
//		}
//	}
//
//	fclose(in_file);
//}
//
//
//void write_new_image_data()
//{
//	int i, j;
//	// Add the path to the name
//	strcat(file_path, name); // images/option/name_option.pgm
//
//	in_file = fopen(file_path, "w");
//	fprintf(in_file, "P2\n");
//	fprintf(in_file, "#Created by Visual Studio - Polycarp\n");
//	fprintf(in_file, "%3d %3d\n", n2 + 1, n2 + 1);
//	fprintf(in_file, "255\n");
//	for (i = p; i <= n1 + p; i++)
//	{
//		for (j = p; j <= n2 + p; j++)
//		{
//			v[i][j] = (int)(u[i][j] * 255.0 + 0.5);
//			fprintf(in_file, "%4d ", v[i][j]);
//		}
//		fprintf(in_file, "\n");
//	}
//	printf("\nSuccessfully saved the new image.\n");
//}
//
///**********************file i/o functions*******************************/
///***********************************************************************/
//
///*gets a pgm image file.*/
//void getpgmfile(char filename[], PGMImage *img, char mode[])
//{
//	char ch;
//	int type;
//
//	in_file = fopen(file_path, mode);
//	tmp_file = fopen(file_path, mode);
//	if (tmp_file == NULL)
//	{
//		fprintf(stderr, "error: unable to open file %s\n\n", file_path);
//		exit(8);
//	}
//
//	printf("\nreading image file: %s\n", filename);
//
//	/*determine pgm image type (only type three can be used)*/
//	ch = getc(tmp_file);
//	if (ch != 'P')
//	{
//		printf("error(1): not valid pgm/ppm file type\n");
//		exit(1);
//	}
//	ch = getc(tmp_file);
//	/*convert the one digit integer currently represented as a character to
//	an integer(48 == '0')*/
//	type = ch - 48;
//	if ((type != 2) && (type != 3) && (type != 5) && (type != 6))
//	{
//		printf("error(2): not valid pgm/ppm file type\n");
//		exit(1);
//	}
//
//	while (getc(tmp_file) != '\n');             /* skip to end of line*/
//
//	while (getc(tmp_file) == '#')              /* skip comment lines */
//	{
//		while (getc(tmp_file) != '\n');          /* skip to end of comment line */
//	}
//
//
//	/*there seems to be a difference between color and b/w.  this line is needed
//	by b/w but doesn't effect color reading...*/
//	fseek(tmp_file, -3, SEEK_CUR);             /* backup *three characters*/
//
//	fscanf(tmp_file, "%d", &((*image).width));
//	fscanf(tmp_file, "%d", &((*image).height));
//	fscanf(tmp_file, "%d", &((*image).maxVal));
//	fclose(tmp_file);
//
//
//
//	if (((*img).width  > MAX) || ((*img).height  > MAX))
//	{
//		printf("\n\n***error - image too big for current image structure***\n\n");
//		exit(1);
//	}
//
//
//	height = (*image).height - 1;
//	width = (*image).width - 1;
//	// write the image data
//	char pom1[5], line[80];
//
//	//reading the headers
//	fgets(pom1, 10, in_file);
//	do {
//		fgets(line, 80, in_file);
//	} while (line[0] == '#');
//	fgets(line, 10, in_file);
//
//	// load appropriate reading mode
//	if (strcmp(mode, "r") == 0)
//	{
//		//read_image_ascii();
//		load_ascii();
//	}
//	else if (strcmp(mode, "rb") == 0)
//	{
//		load_binary();
//	}
//	printf("\ndone reading file.\n");
//}
//
//void printimageinfo(void)
//{
//	printf("\nimage properties:\n");
//	printf("\n file name: %s", (*image).name);
//	printf("\n width  = %d", (*image).width);
//	printf("\n height = %d", (*image).height);
//	printf("\n maxval = %d", (*image).maxVal);
//	printf("\n\n");
//}
//
//
///*Specific to Heat Functions*/
//
////==============================================================================
//
//__global__ void picaod_kernel(unsigned int *dev_v, long size, unsigned int *temp)
//{
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	atomicAdd(&(temp[0]), dev_v[offset]);
//
//}
//
//
//void reflexia(pole1 u)
//{
//	int i, j, k, n11, n22;
//	n11 = n1 + p; n22 = n2 + p;
//	for (i = p; i <= n1 + p; i++)
//		for (k = 0; k<p; k++)
//		{
//			u[i][p - k - 1] = u[i][p + k];
//			u[i][n22 + p + k] = u[i][n22 + p - k - 1];
//		}
//
//	for (j = 0; j <= n2 + 2*p; j++)
//		for (k = 0; k<p; k++)
//		{
//			u[p - k - 1][j] = u[p + k][j];
//			u[n11 + p + k][j] = u[n11 + p - k - 1][j];
//		}
//}
//
////==============================================================================
//
////Kernel functions
//
////======================================
//double gradient(double ux, double uy)
//{
//	double gn = pow(ux, 2) + pow(uy, 2);
//	double grad = 1 / (1.0 + coeff*gn);
//	return grad;
//}
//
////==============================================================================
//__global__ void gauss_seidel_kernel(double *dev_u0, double *dev_u, double *dev_ap, double *dev_ae, double *dev_an, double *dev_aw, double *dev_as)
//{
//	int x = threadIdx.x + (blockIdx.x) * blockDim.x;
//	int y = threadIdx.y + (blockIdx.y) * blockDim.y;
//
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	//Set the dimension - square image expected
//	int DIM = n1 + 16*p + 1;
//
//	int west = offset - p;
//	int east = offset + p;
//
//	if (x == 0) west++;
//	if (x == DIM - p) east--;
//
//	int north = offset - DIM;
//	int south = offset + DIM;
//	if (y == 0) north += DIM;
//	if (y == DIM - p) south -= DIM;
//
//	double sor;
//
//	//Evaluate
//	sor = (dev_aw[offset] * dev_u[west] + dev_ae[offset] * dev_u[east] + dev_as[offset] * dev_u[south] + dev_an[offset] * dev_u[north] + dev_u0[offset] * (h*h / tau)) / dev_ap[offset];
//	dev_u[offset] = dev_u[offset] + omega_s*(sor - dev_u[offset]);
//	if (dev_u[offset] < 0.0) dev_u[offset] = 0.0;
//	if (dev_u[offset] > 1.0) dev_u[offset] = 1.0;
//}
//
////==============================================================================
//void gauss() 
//{
//	double y;
//	int i, j;
//
//	for (i = p; i <= n1 + p; i++)
//	{
//		for (j = p; j <= n2 + p; j++)
//		{
//			y = (aw[i][j] * u[i][j - 1] + ae[i][j] * u[i][j + 1] + as[i][j] * u[i + 1][j] + an[i][j] * u[i - 1][j] + u0[i][j] * (h*h / tau)) / ap[i][j];
//			u[i][j] = u[i][j] + omega_s*(y - u[i][j]);
//		}
//	}
//}
////==============================================================================
//
//void Convolution()
//{
//	int i, j, z;
//	double y, error;
//
//	z = 0;
//	do
//	{
//		z = z + 1;
//		for (i = p; i <= n1 + 1; i++)
//		{
//			for (j = p; j <= n2 + 1; j++)
//			{
//				y = (uq[i + 1][j] + uq[i - 1][j] + uq[i][j + 1] + uq[i][j - 1] + (h*h / sigma)*u0[i][j]) / ((h*h / sigma) + 4.0);
//				uq[i][j] = uq[i][j] + omega_c*(y - uq[i][j]);
//			}
//		}
//
//		error = 0.0;
//		for (i = p; i <= n1 + 1; i++)
//		{
//			for (j = p; j <= n2 + 1; j++)
//			{
//				error += pow(uq[i][j] * ((h*h / sigma) + 4.0) - (uq[i + 1][j] + uq[i - 1][j] + uq[i][j + 1] + uq[i][j - 1] + u0[i][j] * (h*h / sigma)), 2);
//			}
//		}
//	} while ((error>tol) && (z<ni));
//	printf("A number of convolution iterations is %d\n", z);
//	printf("Error is %e\n", error);
//}
////==============================================================================
//void ap_s() 
//{
//	int i, j;
//	double uqx, uqy;
//
//	for (i = p; i <= n1 + 1; i++)
//	{
//		for (j = p; j <= n2 + 1; j++)
//		{
//			uqx = (uq[i][j] - uq[i][j - 1]) / h;
//			uqy = (uq[i - 1][j + 1] - uq[i - 1][j - 1] + uq[i][j + 1] - uq[i][j - 1]) / (4 * h);
//			aw[i][j] = gradient(uqx, uqy);
//
//			uqx = (uq[i][j + 1] - uq[i][j]) / h;
//			uqy = (uq[i][j + 1] - uq[i][j - 1] + uq[i + 1][j + 1] - uq[i + 1][j - 1]) / (4 * h);
//			ae[i][j] = gradient(uqx, uqy);
//
//			uqx = (uq[i + 1][j] - uq[i - 1][j] + uq[i + 1][j - 1] - uq[i - 1][j - 1]) / (4 * h);
//			uqy = (uq[i][j] - uq[i + 1][j]) / h;
//			as[i][j] = gradient(uqx, uqy);
//
//			uqx = (uq[i + 1][j + 1] - uq[i - 1][j + 1] + uq[i + 1][j] - uq[i - 1][j]) / (4 * h);
//			uqy = (uq[i - 1][j] - uq[i][j]) / h;
//			an[i][j] = gradient(uqx, uqy);
//
//			ap[i][j] = (h*h / tau) + aw[i][j] + ae[i][j] + as[i][j] + an[i][j];
//		}
//	}
//}
//
////==============================================================================
//// Main Function
//
//int main()
//{
//	int i,j,l,s;
//	double error_s;
//	/* memory allocation*/
//	name = (char*)malloc(sizeof(PGMImage));
//	name_ending = (char*)malloc(sizeof(PGMImage));
//	file_path = (char*)malloc(sizeof(PGMImage));
//	rmode = (char*)malloc(sizeof(PGMImage));
//
//
//	/*
//	* read in image file. - note: sets our global values, too.
//	* ----------------------------------------------------------------- */
//
//	image = (PGMImage*)malloc(sizeof(PGMImage));
//
//	strcpy(file_path, "images/");
//
//	//prompt for a name without extension
//	printf("enter pgm file name without extension:\n");
//	scanf("%s", &(*image).name);
//	/*
//	* get the mode of the file from the user
//	*/
//	/*printf("enter the mode for the file that you want to read e.g r, rb: ");
//	scanf("%s", rmode);*/
//	/* Manual enter file name and mode*/
//	rmode = "r";
//	strcpy(name, (*image).name);//construct a name of the file
//	sprintf(name_ending, ".pgm");
//	strcat(name, name_ending);
//	strcat(file_path, name);
//	getpgmfile(name, image, rmode);
//	printimageinfo();
//
//	//Get image size
//	int img_size = (n1 + 16 * p + 1)*(n2 + 16 * p + 1);
//
//	/*capture time events gou*/
//	cudaEvent_t start_gpu, stop_gpu;
//
//
//	/*GPU variables*/
//	double *d_u0, *d_u, *d_uq, *d_error, *d_ae, *d_an, *d_aw, *d_as, *d_ap;
//	unsigned int *d_v, *d_tmp;
//
//	/* Important!
//	* Dim % Sqrt(threads) == 0
//	* Target atleast 256 threads
//	*/
//	dim3 blocks((n1 + 16 * p + 1) / 16, (n2 + 16 * p + 1) / 16);
//	dim3 threads(16, 16);
//
//
//	// allocate memory on the GPU for the variables
//	HANDLE_ERROR(cudaMalloc((void **)&d_u0, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_u, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_uq, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_error, img_size * sizeof(double)));
//
//	// Gradients
//	HANDLE_ERROR(cudaMalloc((void **)&d_ae, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_an, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_aw, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_as, img_size * sizeof(double)));
//	// ap
//	HANDLE_ERROR(cudaMalloc((void **)&d_ap, img_size * sizeof(double)));
//	//
//	HANDLE_ERROR(cudaMalloc((void **)&d_tmp, 1 * sizeof(long)));
//	HANDLE_ERROR(cudaMemset(d_tmp, 0, 1 * sizeof(int)));
//
//	HANDLE_ERROR(cudaMalloc((void **)&d_v, img_size * sizeof(int)));
//	HANDLE_ERROR(cudaMemcpy(d_v, v, img_size * sizeof(int), cudaMemcpyHostToDevice));
//
//	// AOD - Function
//	unsigned int tmp[1];
//
//	picaod_kernel << <blocks, threads >> > (d_v, img_size, d_tmp);
//	// Copy tmp from Device to Host
//	HANDLE_ERROR(cudaMemcpy(tmp, d_tmp, 1 * sizeof(int), cudaMemcpyDeviceToHost));
//
//	printf("AOD of the picture before the heat is: %lf \n", tmp[0] / ((double)(n1 + 1)*(n2 + 1)));
//
//	/*Perform PM-Implicit*/
//	partial_error = (double *)malloc(sizeof(double)*img_size);
//
//	reflexia(u);
//	for (l = 1; l <= kk; l++)
//	{
//		HANDLE_ERROR(cudaEventCreate(&start_gpu));
//		HANDLE_ERROR(cudaEventCreate(&stop_gpu));
//		HANDLE_ERROR(cudaEventRecord(start_gpu, 0));
//
//		printf("%d-th step\n", l);
//		/*Begin of Coefficient function*/
//
//		//Copy from u to uq - Can use Memcopy instead
//		memcpy(uq, u, sizeof(u));
//
//		/*Begin of Convolution function*/
//		//Copy from uq to u0
//		memcpy(u0, uq, sizeof(uq));
//		// Reflection uq - CPU function
//		reflexia(uq);
//
//		Convolution();
//		/*Calculate the ap*/
//
//
//		ap_s();
//		// Copy ap,aw,ae,aw,an to device
//		// Copy ap to d_ap
//		HANDLE_ERROR(cudaMemcpy(d_ap, ap, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		// ae,an,aw,as
//		HANDLE_ERROR(cudaMemcpy(d_ae, ae, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		HANDLE_ERROR(cudaMemcpy(d_an, an, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		HANDLE_ERROR(cudaMemcpy(d_aw, aw, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		HANDLE_ERROR(cudaMemcpy(d_as, as, img_size * sizeof(double), cudaMemcpyHostToDevice));
//
//		/*End of Coefficient function*/
//
//		/*Begin of System Equation*/
//		//Copy from u to u0
//		memcpy(u0, u, sizeof(u));
//		// Reflection
//		reflexia(u);
//		// Copy u,u0 to d_u,d_u0
//		HANDLE_ERROR(cudaMemcpy(d_u, u, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		HANDLE_ERROR(cudaMemcpy(d_u0, u0, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		
//		/*Begin do while */
//		s = 0;
//		do
//		{
//			s = s + 1;
//			//Evaluate using the Gauss Seidel method
//			// gauss();
//			// Reflection
//			//reflexia(u);
//			// Copy the reflected u to d_u
//			// HANDLE_ERROR(cudaMemcpy(d_u, u, img_size * sizeof(double), cudaMemcpyHostToDevice));
//
//			gauss_seidel_kernel << <blocks, threads >> > (d_u0, d_u, d_ap, d_ae, d_an, d_aw, d_as);
//			// Use cudaDeviceSynchronize(); until process is done
//			cudaDeviceSynchronize();
//			//Copy back to Host - u for error calculation
//			HANDLE_ERROR(cudaMemcpy(u, d_u, img_size * sizeof(double), cudaMemcpyDeviceToHost));
//
//			// Evaluate error
//
//			// Reflection
//			//reflexia(u);
//
//			//Reset to Zero before next iteration
//			error_s = 0.0;
//
//			for (i = p; i < n1 + 1; i++)
//			{
//				for (j = p; j < n2 + 1; j++)
//				{
//					error_s += pow(ap[i][j] * u[i][j] - ae[i][j] * u[i][j + 1] - as[i][j] * u[i + 1][j] - an[i][j] * u[i - 1][j] - aw[i][j] * u[i][j - 1] - u0[i][j] * (h*h / tau), 2);
//				}
//			}
//			// Copy the reflected u to d_u
//			HANDLE_ERROR(cudaMemcpy(d_u, u, img_size * sizeof(double), cudaMemcpyHostToDevice));
//
//		} while ((error_s > tol) && (s < ni));
//		printf("The number of  system iterations is %d\n", s);
//		printf("error is %e\n", error_s);
//		
//		/* End of System Equation function*/
//		// Use cudaDeviceSynchronize(); until process is done
//		cudaDeviceSynchronize();
//		//Copy back to Host - u
//		HANDLE_ERROR(cudaMemcpy(u, d_u, img_size * sizeof(double), cudaMemcpyDeviceToHost));
//
//		// Reflection
//		reflexia(u);
//
//		// get stop time, and display the timing results
//		HANDLE_ERROR(cudaEventRecord(stop_gpu, 0));
//		HANDLE_ERROR(cudaEventSynchronize(stop_gpu));
//		float   elapsedTime;
//		HANDLE_ERROR(cudaEventElapsedTime(&elapsedTime, start_gpu, stop_gpu));
//		printf("Time to generate the %d-th step:  %3.1f ms\n", l, elapsedTime);
//
//		// Destroy time
//		HANDLE_ERROR(cudaEventDestroy(start_gpu));
//		HANDLE_ERROR(cudaEventDestroy(stop_gpu));
//
//
//		// Write the outputs
//		if ((l%iz) == 0)
//		{
//			strcpy(name, (*image).name);
//			sprintf(name_ending, "_%d_pm.pgm", l);
//			strcat(name, name_ending);
//			strcpy(file_path, "images/modified/pm/semi/");
//			write_new_image_data();
//		}
//
//	}
//
//	// AOD - function
//	HANDLE_ERROR(cudaMemcpy(d_v, v, img_size * sizeof(int), cudaMemcpyHostToDevice));
//
//	// Reset values back to 0
//	HANDLE_ERROR(cudaMemset(d_tmp, 0, 1 * sizeof(int)));
//
//	picaod_kernel << <blocks, threads >> > (d_v, img_size, d_tmp);
//	// Copy tmp from Device to Host
//	HANDLE_ERROR(cudaMemcpy(tmp, d_tmp, 1 * sizeof(int), cudaMemcpyDeviceToHost));
//
//	printf("AOD of the picture before the heat is: %lf \n", tmp[0] / ((double)(n1 + 1)*(n2 + 1)));
//
//
//	fclose(in_file);
//	// free more memory
//	free((void*)name);
//
//	/*gpu free memory*/
//	HANDLE_ERROR(cudaFree(d_u0));
//	HANDLE_ERROR(cudaFree(d_u));
//	HANDLE_ERROR(cudaFree(d_uq));
//	HANDLE_ERROR(cudaFree(d_ap));
//
//	HANDLE_ERROR(cudaFree(d_tmp));
//	HANDLE_ERROR(cudaFree(d_v));
//
//	return 0;
//
//}