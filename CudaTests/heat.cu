
#include <hip/hip_runtime.h>
//#pragma warning(disable:4996)
//
//
//#include <device_functions.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include "common\book.h"
//
//
//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h> // math functions
//#include <string.h> // functions such as strcmp
//#include <iostream>
//#include <time.h>
//
//
///* a header file with common structure to for pgm file*/
//#include "src/pgm.h" // common header for common pgm functions
//
//
//#define MAX_FILE_LENGTH 256
//
////nucleia.pgm ascii
//#define  n1 511  //height of the image -1-rows
//#define  n2 511  //width of the image -1-columns
//
//
///* Heat Definitions*/
//#define kk     10        /* a number of the diffusion steps */
//#define  iz     1         /* density of writing */
//#define  p     1
//
//
//typedef  double pole1[n1 + 16 * p + 1][n2 + 16 * p + 1];
//typedef  int    pole2[n1 + 16 * p + 1][n2 + 16 * p + 1];
//
////pole--array
//static pole1 u, u0;
//static pole2 v;
//
//
//PGMImage *image; // image struct with the propertes: width, height, maxvalue
//char *name, *name_ending, *name_beg, *file_path, *rmode;
//int height = 0, width = 0; //height, width for input image
//int option, prd, off;
//FILE *in_file, *tmp_file;
//
///*functions for reading and writing*/
///*
//* reads the image data into a 2d array
//* img_array contains the image data
//*/
//
//void load_ascii()
//{
//	int ll, i, j;
//	//reading the image data - ascii_format
//	for (i = p; i <= height + p; i++)
//	{
//		for (j = p; j <= width + p; j++)
//		{
//			fscanf(in_file, "%d", &ll);
//			v[i][j] = ll;
//			u[i][j] = ll / 255.;
//		}
//	}
//	fclose(in_file);
//}
//
//void load_binary()
//{
//	int ll, i, j;
//	//reading the image data- binary format
//	for (i = p; i <= height + p; i++)
//	{
//		for (j = p; j <= width + p; j++)
//		{
//			ll = getc(in_file);
//			v[i][j] = ll;
//			u[i][j] = ll / 255.;
//		}
//	}
//
//	fclose(in_file);
//}
//
//
//void write_new_image_data()
//{
//	int i, j;
//	// Add the path to the name
//	strcat(file_path, name); // images/option/name_option.pgm
//
//	in_file = fopen(file_path, "w");
//	fprintf(in_file, "P2\n");
//	fprintf(in_file, "#Created by Visual Studio - Polycarp\n");
//	fprintf(in_file, "%3d %3d\n", n2 + 1, n2 + 1);
//	fprintf(in_file, "255\n");
//	for (i = p; i <= n1 + p; i++)
//	{
//		for (j = p; j <= n2 + p; j++)
//		{
//			v[i][j] = (int)(u[i][j] * 255.0 + 0.5);
//			fprintf(in_file, "%4d ", v[i][j]);
//		}
//		fprintf(in_file, "\n");
//	}
//	fclose(in_file);
//	printf("\nSuccessfully saved the new image.\n");
//}
//
///**********************file i/o functions*******************************/
///***********************************************************************/
//
///*gets a pgm image file.*/
//void getpgmfile(char filename[], PGMImage *img, char mode[])
//{
//	char ch;
//	int type;
//
//	in_file = fopen(file_path, mode);
//	tmp_file = fopen(file_path, mode);
//	if (tmp_file == NULL)
//	{
//		fprintf(stderr, "error: unable to open file %s\n\n", file_path);
//		exit(8);
//	}
//
//	printf("\nreading image file: %s\n", filename);
//
//	/*determine pgm image type (only type three can be used)*/
//	ch = getc(tmp_file);
//	if (ch != 'P')
//	{
//		printf("error(1): not valid pgm/ppm file type\n");
//		exit(1);
//	}
//	ch = getc(tmp_file);
//	/*convert the one digit integer currently represented as a character to
//	an integer(48 == '0')*/
//	type = ch - 48;
//	if ((type != 2) && (type != 3) && (type != 5) && (type != 6))
//	{
//		printf("error(2): not valid pgm/ppm file type\n");
//		exit(1);
//	}
//
//	while (getc(tmp_file) != '\n');             /* skip to end of line*/
//
//	while (getc(tmp_file) == '#')              /* skip comment lines */
//	{
//		while (getc(tmp_file) != '\n');          /* skip to end of comment line */
//	}
//
//
//	/*there seems to be a difference between color and b/w.  this line is needed
//	by b/w but doesn't effect color reading...*/
//	fseek(tmp_file, -3, SEEK_CUR);             /* backup *three characters*/
//
//	fscanf(tmp_file, "%d", &((*image).width));
//	fscanf(tmp_file, "%d", &((*image).height));
//	fscanf(tmp_file, "%d", &((*image).maxVal));
//	fclose(tmp_file);
//
//
//
//	if (((*img).width  > MAX) || ((*img).height  > MAX))
//	{
//		printf("\n\n***error - image too big for current image structure***\n\n");
//		exit(1);
//	}
//
//
//	height = (*image).height - 1;
//	width = (*image).width - 1;
//	// write the image data
//	char pom1[5], line[80];
//
//	//reading the headers
//	fgets(pom1, 10, in_file);
//	do {
//		fgets(line, 80, in_file);
//	} while (line[0] == '#');
//	fgets(line, 10, in_file);
//
//	// load appropriate reading mode
//	if (strcmp(mode, "r") == 0)
//	{
//		//read_image_ascii();
//		load_ascii();
//	}
//	else if (strcmp(mode, "rb") == 0)
//	{
//		load_binary();
//	}
//	printf("\ndone reading file.\n");
//}
//
//void printimageinfo(void)
//{
//	printf("\nimage properties:\n");
//	printf("\n file name: %s", (*image).name);
//	printf("\n width  = %d", (*image).width);
//	printf("\n height = %d", (*image).height);
//	printf("\n maxval = %d", (*image).maxVal);
//	printf("\n\n");
//}
//
//
///*Specific to Heat Functions*/
//
////==============================================================================
//
//__global__ void picaod_kernel(unsigned int *dev_v, long size, unsigned int *temp) 
//{
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	atomicAdd(&(temp[0]), dev_v[offset]);
//
//}
//
//
///* Replaced by Device Function*/
//
////double picaod()
////{
////	int i, j;
////	int tmp;
////	tmp = 0;
////	for (i = p; i <= n1 + p; i++)
////		for (j = p; j <= n2 + p; j++)
////		{
////			tmp += v[i][j];
////		}
////	return tmp / ((double)(n1 + 1)*(n2 + 1));
////}
////------------------------------------------------------------------------------------
//
//void reflexia(pole1 u)
//{
//	int i, j, k, n11, n22;
//	n11 = n1 + 1; n22 = n2 + 1;
//	for (i = p; i <= n1 + p; i++)
//		for (k = 0; k<p; k++)
//		{
//			u[i][p - k - 1] = u[i][p + k];
//			u[i][n22 + p + k] = u[i][n22 + p - k - 1];
//		}
//
//	for (j = 0; j <= n2 + 2 * p; j++)
//		for (k = 0; k<p; k++)
//		{
//			u[p - k - 1][j] = u[p + k][j];
//			u[n11 + p + k][j] = u[n11 + p - k - 1][j];
//		}
//}
//
////==============================================================================
//
//
//
////Kernel functions
//
////======================================
//__global__ void heat_explicit(double *dev_u0, double *dev_u, double tau, int h)
//{
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	int offset = x + y * blockDim.x * gridDim.x;
//
//	//Set the dimension - square image expected
//	int DIM = n1 + 16 * p + 1;
//
//	int left = offset - p;
//	int right = offset + p;
//
//	if (x == 0) left++;
//	if (x == DIM -p) right--;
//
//	int top = offset - DIM;
//	int bottom = offset + DIM;
//	if (y == 0) top += DIM;
//	if (y == DIM - p) bottom -= DIM;
//
//	dev_u[offset] = (1 - 4*(tau / (h*h)))*dev_u0[offset] + (tau / (h*h))*(dev_u0[top] + dev_u0[bottom] + dev_u0[left] + dev_u0[right]);
//}
//
//// Main Function
//
//int main()
//{
//	int i;
//	/* memory allocation*/
//	name = (char*)malloc(sizeof(PGMImage));
//	name_ending = (char*)malloc(sizeof(PGMImage));
//	file_path = (char*)malloc(sizeof(PGMImage));
//	rmode = (char*)malloc(sizeof(PGMImage));
//
//
//	/*
//	* read in image file. - note: sets our global values, too.
//	* ----------------------------------------------------------------- */
//
//	image = (PGMImage*)malloc(sizeof(PGMImage));
//
//	strcpy(file_path, "images/");
//
//	//prompt for a name without extension
//	printf("enter pgm file name without extension:\n");
//	scanf("%s", &(*image).name);
//	/*
//	* get the mode of the file from the user
//	*/
//	printf("enter the mode for the file that you want to read e.g r, rb: ");
//	scanf("%s", rmode);
//	strcpy(name, (*image).name);//construct a name of the file
//	sprintf(name_ending, ".pgm");
//	strcat(name, name_ending);
//	strcat(file_path, name);
//	getpgmfile(name, image, rmode);
//	printimageinfo();
//
//	//Get image size
//	int img_size = (n1 + 16 * p + 1)*(n2 + 16 * p + 1);
//
//	/*capture time events gou*/
//	cudaEvent_t start_gpu, stop_gpu;
//	
//
//	/*GPU variables*/
//	double *d_u0, *d_u;
//	unsigned int *d_v, *d_tmp;
//
//	/* Important!
//	* Dim % Sqrt(threads) == 0
//	* Target atleast 256 threads
//	*/
//	dim3 blocks((n1 + 16 * p + 1)/16, (n2 + 16 * p + 1) / 16);
//	dim3 threads(16, 16);
//	
//
//	// allocate memory on the GPU for the variables
//	HANDLE_ERROR(cudaMalloc((void **)&d_u0, img_size * sizeof(double)));
//	HANDLE_ERROR(cudaMalloc((void **)&d_u, img_size * sizeof(double)));
//	//
//	HANDLE_ERROR(cudaMalloc((void **)&d_tmp, 1 * sizeof(long)));
//	HANDLE_ERROR(cudaMemset(d_tmp, 0, 1 * sizeof(int)));
//
//	HANDLE_ERROR(cudaMalloc((void **)&d_v, img_size * sizeof(int)));
//	HANDLE_ERROR(cudaMemcpy(d_v, v, img_size * sizeof(int), cudaMemcpyHostToDevice));
//
//	// AOD - Function
//	unsigned int tmp[1];
//
//	picaod_kernel << <blocks, threads >> > (d_v, img_size, d_tmp);
//	// Copy tmp from Device to Host
//	HANDLE_ERROR(cudaMemcpy(tmp, d_tmp, 1 * sizeof(int), cudaMemcpyDeviceToHost));
//
//	printf("AOD of the picture before the heat is: %lf \n", tmp[0] / ((double)(n1 + 1)*(n2 + 1)));
//
//	/*Perform Heat Convolution*/
//	double tau = 0.25;
//	int h = 1;
//	reflexia(u);
//	for (i = 1; i <= kk; i++)
//	{
//		HANDLE_ERROR(cudaEventCreate(&start_gpu));
//		HANDLE_ERROR(cudaEventCreate(&stop_gpu));
//		HANDLE_ERROR(cudaEventRecord(start_gpu, 0));
//
//		printf("%d-th step\n", i);
//		//Copy from u to u0 - Can use Memcopy instead
//		memcpy(u0,u, sizeof(u));
//		// Reflection u0 - CPU function
//		reflexia(u0);
//		//Copy u0 to d_u0
//		HANDLE_ERROR(cudaMemcpy(d_u0, u0, img_size * sizeof(double), cudaMemcpyHostToDevice));
//		// Perform heat explicit
//		heat_explicit << <blocks, threads >> > (d_u0, d_u, tau, h);
//		// Use cudaDeviceSynchronize(); until process is done
//		//cudaDeviceSynchronize();
//		//Copy back to Host - u
//		HANDLE_ERROR(cudaMemcpy(u, d_u, img_size * sizeof(double), cudaMemcpyDeviceToHost));
//
//		// get stop time, and display the timing results
//		HANDLE_ERROR(cudaEventRecord(stop_gpu, 0));
//		HANDLE_ERROR(cudaEventSynchronize(stop_gpu));
//		float   elapsedTime;
//		HANDLE_ERROR(cudaEventElapsedTime(&elapsedTime, start_gpu, stop_gpu));
//		printf("Time to generate the %d-th step:  %3.1f ms\n", i,elapsedTime);
//
//		// Destroy time
//		HANDLE_ERROR(cudaEventDestroy(start_gpu));
//		HANDLE_ERROR(cudaEventDestroy(stop_gpu));
//
//
//		// Write the outputs
//		if ((i%iz) == 0)
//		{
//			strcpy(name, (*image).name);
//			sprintf(name_ending, "_%d_lhe.pgm", i);
//			strcat(name, name_ending);
//			strcpy(file_path, "images/modified/explicit/heat/");
//			write_new_image_data();
//		}
//	
//	}
//
//	// AOD - function
//	HANDLE_ERROR(cudaMemcpy(d_v, v, img_size * sizeof(int), cudaMemcpyHostToDevice));
//
//	// Reset values back to 0
//	HANDLE_ERROR(cudaMemset(d_tmp, 0, 1 * sizeof(int)));
//
//	picaod_kernel << <blocks, threads >> > (d_v, img_size, d_tmp);
//	// Copy tmp from Device to Host
//	HANDLE_ERROR(cudaMemcpy(tmp, d_tmp, 1 * sizeof(int), cudaMemcpyDeviceToHost));
//
//	printf("AOD of the picture before the heat is: %lf \n", tmp[0] / ((double)(n1 + 1)*(n2 + 1)));
//
//	
//	fclose(in_file);
//	// free more memory
//	free((void*)name);
//
//	/*gpu free memory*/
//	HANDLE_ERROR(cudaFree(d_u0));
//	HANDLE_ERROR(cudaFree(d_u));
//
//	HANDLE_ERROR(cudaFree(d_tmp));
//	HANDLE_ERROR(cudaFree(d_v));
//
//	return 0;
//
//}