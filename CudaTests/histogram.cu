
#include <hip/hip_runtime.h>
//#pragma warning(disable:4996)
//
//
//#include <device_functions.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include "common\book.h"
//
//
//#include <stdio.h>
//#include <stdlib.h>
//#include <math.h> // math functions
//#include <string.h> // functions such as strcmp
//#include <iostream>
//#include <time.h>
//
//
///* a header file with common structure to for pgm file*/
//#include "src/pgm.h" // common header for common pgm functions
//
//
//#define MAX_FILE_LENGTH 256
//
//#define p 0
//
////memb1.pgm -binary
//#define  n1 199   //height of the image -1-rows
//#define  n2 199  //width of the image -1-collumns
//
//
//typedef  int    array2d[n1 + 2 * p + 1][n2 + 2 * p + 1];//def new type array2d
//														//we define new type array2d
//static array2d v;//array for image
//
//PGMImage *image; // image struct with the propertes: width, height, maxvalue
//char *name, *name_ending, *name_beg, *file_path, *rmode;
//int height = 0, width = 0; //height, width for input image
//int option, prd, off;
//FILE *in_file, *tmp_file;
//
///*functions for reading and writing*/
///*
//* reads the image data into a 2d array
//* img_array contains the image data
//*/
//
//void load_ascii()
//{
//	int ll, i, j;
//	//reading the image data - ascii_format
//	for (i = p; i <= height + p; i++)
//	{
//		for (j = p; j <= width + p; j++)
//		{
//			fscanf(in_file, "%d", &ll);
//			v[i][j] = ll;
//		}
//	}
//	fclose(in_file);
//}
//
//void load_binary()
//{
//	int ll, i, j;
//	//reading the image data- binary format
//	for (i = p; i <= height + p; i++)
//	{
//		for (j = p; j <= width + p; j++)
//		{
//			ll = getc(in_file);
//			v[i][j] = ll;
//
//		}
//	}
//
//	fclose(in_file);
//}
//
//
//void write_new_image_data(array2d v)
//{
//	int i, j;
//	// Add the path to the name
//	strcat(file_path, name); // images/option/name_option.pgm
//
//	in_file = fopen(file_path, "wb");
//	fprintf(in_file, "P5\n");
//	fprintf(in_file, "#Created by Visual Studio - Polycarp\n");
//	fprintf(in_file, "%3d %3d\n", n2 + 1, n2 + 1);
//	fprintf(in_file, "255\n");
//	for (i = p; i <= n1 + p; i++)
//		for (j = p; j <= n2 + p; j++)
//		{
//			putc(v[i][j], in_file);
//		}
//	printf("\nSuccessfully saved the new image.\n");
//}
//
///**********************file i/o functions*******************************/
///***********************************************************************/
//
///*gets a pgm image file.*/
//void getpgmfile(char filename[], PGMImage *img, char mode[])
//{
//	char ch;
//	int type;
//
//	in_file = fopen(file_path, mode);
//	tmp_file = fopen(file_path, mode);
//	if (tmp_file == NULL)
//	{
//		fprintf(stderr, "error: unable to open file %s\n\n", file_path);
//		exit(8);
//	}
//
//	printf("\nreading image file: %s\n", filename);
//
//	/*determine pgm image type (only type three can be used)*/
//	ch = getc(tmp_file);
//	if (ch != 'P')
//	{
//		printf("error(1): not valid pgm/ppm file type\n");
//		exit(1);
//	}
//	ch = getc(tmp_file);
//	/*convert the one digit integer currently represented as a character to
//	an integer(48 == '0')*/
//	type = ch - 48;
//	if ((type != 2) && (type != 3) && (type != 5) && (type != 6))
//	{
//		printf("error(2): not valid pgm/ppm file type\n");
//		exit(1);
//	}
//
//	while (getc(tmp_file) != '\n');             /* skip to end of line*/
//
//	while (getc(tmp_file) == '#')              /* skip comment lines */
//	{
//		while (getc(tmp_file) != '\n');          /* skip to end of comment line */
//	}
//
//
//	/*there seems to be a difference between color and b/w.  this line is needed
//	by b/w but doesn't effect color reading...*/
//	fseek(tmp_file, -3, SEEK_CUR);             /* backup *three characters*/
//
//	fscanf(tmp_file, "%d", &((*image).width));
//	fscanf(tmp_file, "%d", &((*image).height));
//	fscanf(tmp_file, "%d", &((*image).maxVal));
//	fclose(tmp_file);
//
//
//
//	if (((*img).width  > MAX) || ((*img).height  > MAX))
//	{
//		printf("\n\n***error - image too big for current image structure***\n\n");
//		exit(1);
//	}
//
//
//	height = (*image).height - 1;
//	width = (*image).width - 1;
//	// write the image data
//	char pom1[5], line[80];
//
//	//reading the headers
//	fgets(pom1, 10, in_file);
//	do {
//		fgets(line, 80, in_file);
//	} while (line[0] == '#');
//	fgets(line, 10, in_file);
//
//	// load appropriate reading mode
//	if (strcmp(mode, "r") == 0)
//	{
//		//read_image_ascii();
//		load_ascii();
//	}
//	else if (strcmp(mode, "rb") == 0)
//	{
//		load_binary();
//	}
//	printf("\ndone reading file.\n");
//}
//
//void printimageinfo(void)
//{
//	printf("\nimage properties:\n");
//	printf("\n file name: %s", (*image).name);
//	printf("\n width  = %d", (*image).width);
//	printf("\n height = %d", (*image).height);
//	printf("\n maxval = %d", (*image).maxVal);
//	printf("\n\n");
//}
//
//
///*Specific to Histogram Functions*/
//__global__ void histo_kernel(unsigned int *image, long size, unsigned int *histo)
//{
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	int stride = blockDim.x * gridDim.x;
//
//	while (i < size) {
//		atomicAdd( &(histo[image[i]]), 1);
//		i += stride;
//	}
//
//	
//}
//
//
//
//
//int main() 
//{
//	int i,j;
//	/*capture time events gou*/
//	//gpu
//	cudaEvent_t start_gpu, stop_gpu;
//	HANDLE_ERROR(cudaEventCreate(&start_gpu));
//	HANDLE_ERROR(cudaEventCreate(&stop_gpu));
//	HANDLE_ERROR(cudaEventRecord(start_gpu, 0));
//
//	/*end capture*/
//	/* memory allocation*/
//	name = (char*)malloc(sizeof(PGMImage));
//	name_ending = (char*)malloc(sizeof(PGMImage));
//	file_path = (char*)malloc(sizeof(PGMImage));
//	rmode = (char*)malloc(sizeof(PGMImage));
//
//
//	/*
//	* read in image file. - note: sets our global values, too.
//	* ----------------------------------------------------------------- */
//
//	image = (PGMImage*)malloc(sizeof(PGMImage));
//
//	strcpy(file_path, "images/");
//
//	//prompt for a name without extension
//	printf("enter pgm file name without extension:\n");
//	scanf("%s", &(*image).name);
//	/*
//	* get the mode of the file from the user
//	*/
//	printf("enter the mode for the file that you want to read e.g r, rb: ");
//	scanf("%s", rmode);
//	strcpy(name, (*image).name);//construct a name of the file
//	sprintf(name_ending, ".pgm");
//	strcat(name, name_ending);
//	strcat(file_path, name);
//	getpgmfile(name, image, rmode);
//	printimageinfo();
//
//	//Get image size
//	int img_size = (n1+2*p + 1)*(n2+2*p + 1);
//
//
//	/*GPU variables*/
//	unsigned int *dev_image;
//	unsigned int *dev_histogram;
//
//	double *d_hist;
//	HANDLE_ERROR(cudaMalloc((void**)&d_hist, 256 * sizeof(double)));
//
//
//
//	// allocate memory on the GPU for the file's data
//	HANDLE_ERROR(cudaMalloc((void **)&dev_image, img_size * sizeof(int)));
//	HANDLE_ERROR(cudaMemcpy(dev_image, v, img_size * sizeof(int), cudaMemcpyHostToDevice));
//
//
//	HANDLE_ERROR(cudaMalloc((void**)&dev_histogram,256 * sizeof(long)));
//	HANDLE_ERROR(cudaMemset(dev_histogram, 0,256 * sizeof(int)));
//
//
//
//	// GEt the Device property and Processor count
//	cudaDeviceProp  prop;
//	HANDLE_ERROR(cudaGetDeviceProperties(&prop, 0));
//	int blocks = prop.multiProcessorCount;
//
//
//	histo_kernel << <blocks * 2, 256 >> > (dev_image, img_size, dev_histogram);
//	unsigned int    histogram[256];
//
//	HANDLE_ERROR(cudaMemcpy(histogram, dev_histogram, 256 * sizeof(int), cudaMemcpyDeviceToHost));
//
//
//
//	// get stop time, and display the timing results
//	HANDLE_ERROR(cudaEventRecord(stop_gpu, 0));
//	HANDLE_ERROR(cudaEventSynchronize(stop_gpu));
//	float   elapsedTime;
//	HANDLE_ERROR(cudaEventElapsedTime(&elapsedTime, start_gpu, stop_gpu));
//	printf("Time to generate:  %3.1f ms\n", elapsedTime);
//
//	long histoCount = 0;
//	for (int i = 0; i<256; i++) {
//		histoCount += histogram[i];
//	}
//
//	printf("h[0]:%d \n", histogram[0]);
//	printf("Histogram Sum:  %ld\n", histoCount);
//
//
//	
//
//	// Normalization
//	double hist_d[256];
//	for (int i = 0; i<256; i++) {
//		hist_d[i] = histogram[i]/(double)img_size;
//	}
//	printf("h[0]/n*m:%lf \n\n", hist_d[0]);
//	
//
//	fclose(in_file);
//	// free more memory
//	free((void*)name);
//	// Destroy time
//	HANDLE_ERROR(cudaEventDestroy(start_gpu));
//	HANDLE_ERROR(cudaEventDestroy(stop_gpu));
//	/*gpu free memory*/
//	cudaFree(d_hist);
//	cudaFree(dev_image);
//	cudaFree(dev_histogram);
//
//	return 0;
//
//}